#include "hip/hip_runtime.h"
﻿//CUDA Assessment Project for Etecube by Yasin Cesur
//
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <stdio.h>
#include <iostream>
#include <queue> //queue usage
#include <chrono> //delay usage
#include <thread> //delay usage
#define SIZE 100 //runway lenght, thread count, athlete count
using namespace std;

//Athlete Class   ***other infos may be filled from a file easily as long as ID match***
class Athlete {
    //Members
    public: 
        int id;
        int run = 0;
        string name;
        string gender;
        int age;
        string nation;
    //Fuctions
};

//function for filling random numbers between 1 and 5 to an array
void randomGeneration(int* b) {
    for (int i = 0; i < SIZE; i++)
        b[i] = 1 + (rand() % 5);
}

//function for adding random meters to athlete runs **by GPU**
__global__ void randomMeters(int* a, int* b, int* c)
{
    int i = threadIdx.x; //using threads parallel

    c[i] = a[i] + b[i];
}

int main()
{
    //timer 
    using namespace std::this_thread;
    using namespace std::chrono;
    int runFinish = 0; //variable for checking race finished or not
    int anyOneFinish = 0; //variable for checking if anyone finished or not

    srand((unsigned)time(NULL)); //providing a seed value for random number generation

    int * a, *b, *c;

    queue<int> placementQ; //placement queue

    //creating 100 Athlete Objects
    Athlete TheHundred[SIZE]{}; //Lexa <3

    //memory space
    hipMallocManaged(&a, SIZE * sizeof(int));
    hipMallocManaged(&b, SIZE * sizeof(int));
    hipMallocManaged(&c, SIZE * sizeof(int));
    
    //setting ID's of athletes
    for (int i = 0; i < SIZE; i++) {
        TheHundred[i].id = i + 1;
    }

    cout << "Race is started!\n\n";

    //a loop that will work until race is finished, with 1 second delay
    while (runFinish != 1) 
    {
        randomGeneration(b); //generating random numbers

        //random meters/second 
        randomMeters << <1, SIZE >> > (a, b, c); //calculated by GPU with 1 Block, 100 Threads
        hipDeviceSynchronize();

        //transferring meters to athlete objects
        for (int j = 0; j < SIZE; j++) {
            TheHundred[j].run += c[j];

            //if any athlete finishes race print current run only once
            if (TheHundred[j].run >= 100 && anyOneFinish != 1) {
                cout << "The athlete whose ID is " << TheHundred[j].id << ", finished first!\n\n";
                cout << "\tID\t**\tRUN\n";
                for (int k = 0; k < SIZE; k++) //print all current situation
                    cout << "\t" << TheHundred[k].id << "\t**\t" << TheHundred[k].run << "\n";
                anyOneFinish = 1;
            }

            //fill race placement
            if (TheHundred[j].run >= 100) {
                placementQ.push(TheHundred[j].id); //put athlete ID to placement if finished
                TheHundred[j].run = -100; //reduce run value to avoid same ID push
            }
        }
            
        //finish the run if each athlete finished race
        if (placementQ.size() >= 100)
            runFinish = 1;

        sleep_for(seconds(1)); //1 second delay
    }

    //print the placement
    int place = 1;
    cout << "\n\n\tPLACE\t**\tID\n";
    while (!placementQ.empty()) {
        cout << "\t" << place << "\t**\t" << placementQ.front();
        placementQ.pop();
        cout << "\n";

        place++;
    }

    //free memory space
    hipFree(a);
    hipFree(b);
    hipFree(c);
   
    return 0;
}